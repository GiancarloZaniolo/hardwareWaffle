#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "waffle.h"

// #include <unistd.h>
// #include <stdlib.h>
// #include <iomanip>


#define DIM_OF_BLOCK 512

// Constants for use on GPU
struct GlobalConstants {
  int rows;
  int cols;
  int *input_mat;
  int pair_list_len;
  int *pair_list;
  int *all_pair_best;
};

__constant__ GlobalConstants cu_glob_const_params;


// **************************************** //
// ************** OUR KERNELS ************* //
// **************************************** //


// Create the prefix sum array
__global__ void kernel_sum_prefix(int max_row) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if(row >= max_row) {
    return;
  }

  int cols = cu_glob_const_params.cols;
  int *input_mat = cu_glob_const_params.input_mat;

  // Sum prefixes for my row:
  int curr = input_mat[row * cols];
  for(int col = 1; col < cols; col++) {
    curr = curr + input_mat[row * cols + col];
    input_mat[row * cols + col] = curr;
  }

}


__device__ int getRowCuda(int idx) {
  // I hope we don't get weird floating point rounding error lol
    return (-1 + ((int)sqrtf((float)(1+8*idx)))) / 2;
}

__device__ int rowToIdxCuda(int row) {
    return ((row * (row + 1)) / 2);
}


// Generage MCSS values for all of the pairs
__global__ void find_best_each_combo(int max_idx) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= max_idx) {
    return;
  }

  int rows = cu_glob_const_params.rows;
  int cols = cu_glob_const_params.cols;
  int *input_mat = cu_glob_const_params.input_mat;
  int *all_pair_best = cu_glob_const_params.all_pair_best;


  // Calculate based on idx
  int temp = getRowCuda(idx);
  int temp2 = rowToIdxCuda(temp);
  int start_col = (cols - 1) - temp - 1;
  int end_col = (cols - 1) - (idx - temp2);


  // Calculate mcss as you go

  // Maybe this can be registers lol
  int best[] = {-1, -1, -1};
  int curr[] = {0, 0, -1};
  for(int row = 0; row < rows; row++) {
    int this_val = input_mat[row * cols + end_col] - input_mat[row * cols + start_col];

    curr[0] += this_val;
    curr[2] = row;
    if(curr[0] < 0) {
      curr[0] = 0;
      curr[1] = row+1;
      curr[2] = row+1;
    } else if(curr[0] > best[0]) {
      best[0] = curr[0];
      best[1] = curr[1];
      best[2] = curr[2];
    }
  }
  all_pair_best[idx * 3] = best[0];
  all_pair_best[idx * 3 + 1] = best[1];
  all_pair_best[idx * 3 + 2] = best[2];
}


// For debugging purposes
__global__ void debug_vars() {
  int pair_list_len = cu_glob_const_params.pair_list_len;
  int *pair_list = cu_glob_const_params.pair_list;

  printf("CUDA PAIR LIST\n");
  for(int i = 0; i < (pair_list_len / 2); i++) {
    printf("(%d,(%d,%d)) ",i,pair_list[i*2],pair_list[i*2+1]);
  }
  printf("\n");
}


// **************************************** //
// ************ Class Functions *********** //
// **************************************** //


void WaffleSolver::setup(std::string input_filename) {

  std::ifstream fin(input_filename);

  if(!fin) {
    std::cerr << "Unable to open file: " << input_filename << ".\n";
  }

  fin >> rows >> cols;

  cols = cols + 1;

  input_mat = new int[rows * cols];

  for(int i = 0; i < rows; i++) {
    input_mat[i * cols] = 0;
    for(int j = 1; j < cols; j++) {
      fin >> input_mat[i * cols + j];
    }
  }

  fin.close();

  pair_list_len = ((cols - 1) * cols)/2;

  all_pair_best = new int[(pair_list_len * 3)];
}


void WaffleSolver::cuda_malloc_memcpy() {
  hipMalloc(&cuda_device_input_mat, sizeof(int) * rows * cols);
  hipMalloc(&cuda_device_all_pair_best, sizeof(int) * ((pair_list_len) * 3));

  hipMemcpy(cuda_device_input_mat, input_mat, sizeof(int) * rows * cols, 
    hipMemcpyHostToDevice);

  GlobalConstants params;
  params.rows = rows;
  params.cols = cols;
  params.input_mat = cuda_device_input_mat;
  params.pair_list_len = pair_list_len;
  params.pair_list = cuda_device_pair_list;
  params.all_pair_best = cuda_device_all_pair_best;

  hipMemcpyToSymbol(HIP_SYMBOL(cu_glob_const_params), &params, sizeof(GlobalConstants));
}

// Basic helpers for solve()
int getRow(int idx) {
  // I hope we don't get weird floating point rounding error lol
    return (-1 + ((int)sqrtf((float)(1+8*idx)))) / 2;
}

int rowToIdx(int row) {
    return ((row * (row + 1)) / 2);
}

int WaffleSolver::solve() {
  
  dim3 block_dim_prefix(DIM_OF_BLOCK);
  int grid_dim_amnt_prefix = (rows + DIM_OF_BLOCK - 1) / DIM_OF_BLOCK;
  dim3 grid_dim_prefix(grid_dim_amnt_prefix);
  kernel_sum_prefix<<<grid_dim_prefix, block_dim_prefix>>>(rows);

  dim3 block_dim_best_combo(DIM_OF_BLOCK);
  int grid_dim_amnt_best_combo = (pair_list_len + DIM_OF_BLOCK - 1) / DIM_OF_BLOCK;
  dim3 grid_dim_best_combo(grid_dim_amnt_best_combo);
  find_best_each_combo<<<grid_dim_amnt_best_combo, block_dim_best_combo>>>(pair_list_len);
  
  // Do reduction on uniprocessor, may be faster than kernel launch
  hipMemcpy(all_pair_best, cuda_device_all_pair_best, sizeof(int) * pair_list_len * 3, hipMemcpyDeviceToHost);

  int best_val = -1;
  int best_idx = -1;
  for(int i = 0; i < pair_list_len; i++) {
    if(all_pair_best[i * 3] > best_val) {
      best_val = all_pair_best[i * 3];
      best_idx = i;
    }
  }

  // Calculate columns from index
  int temp = getRow(best_idx);
  int temp2 = rowToIdx(temp);
  int start_col = (cols - 1) - temp - 1;
  int end_col = (cols - 1) - (best_idx - temp2);
  
  return best_val;
}